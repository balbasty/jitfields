#include "hip/hip_runtime.h"
/* TODO
 * - check if using an inner loop across batch elements is more efficient
 *   (we currently use an outer loop, so we recompute indices many times)
 */

#include "cuda_switch.h"
#include "spline.h"
#include "bounds.h"
#include "batch.h"
#include "resize.h"

using namespace std;
using namespace jf;
using namespace jf::resize;

template <spline::type IX, bound::type BX,
          typename scalar_t, typename offset_t>
__global__ void kernel1d(scalar_t * out, scalar_t * inp, int ndim,
                         scalar_t shift, const scalar_t * scale,
                         const offset_t * size_out,
                         const offset_t * size_inp,
                         const offset_t * stride_out,
                         const offset_t * stride_inp)
{
    offset_t index = threadIdx.x + blockIdx.x * blockDim.x;
    offset_t nthreads = prod(size_out, ndim);

    for (offset_t i=index; index < nthreads;
         index += blockDim.x * gridDim.x, i=index)
    {
        offset_t x;
        offset_t batch_offset = index2offset_1d(i, ndim, size_out, stride_inp, x);
        offset_t out_offset = index2offset(i, ndim, size_out, stride_out);

        Multiscale<one, IX, BX>::resize(out + out_offset, inp + batch_offset,
                                        x, size_inp[ndim-1], stride_inp[ndim-1],
                                        scale[ndim-1], shift);
    }
}

template <spline::type IX, bound::type BX,
          spline::type IY, bound::type BY,
          typename scalar_t, typename offset_t>
__global__ void kernel2d(scalar_t * out, scalar_t * inp, int ndim,
                         scalar_t shift, const scalar_t * scale,
                         const offset_t * size_out,
                         const offset_t * size_inp,
                         const offset_t * stride_out,
                         const offset_t * stride_inp)
{
    offset_t index = threadIdx.x + blockIdx.x * blockDim.x;
    offset_t nthreads = prod(size_out, ndim);

    for (offset_t i=index; index < nthreads;
         index += blockDim.x * gridDim.x, i=index)
    {
        offset_t x, y;
        offset_t batch_offset = index2offset_2d(i, ndim, size_out, stride_inp, x, y);
        offset_t out_offset = index2offset(i, ndim, size_out, stride_out);

        Multiscale<two, IX, BX, IY, BY>::resize(
            out + out_offset, inp + batch_offset,
            x, size_inp[ndim-2], stride_inp[ndim-2], scale[ndim-2],
            y, size_inp[ndim-1], stride_inp[ndim-1], scale[ndim-1],
            shift);
    }
}

template <spline::type IX, bound::type BX,
          spline::type IY, bound::type BY,
          spline::type IZ, bound::type BZ,
          typename scalar_t, typename offset_t>
__global__ void kernel3d(scalar_t * out, scalar_t * inp, int ndim,
                         scalar_t shift, const scalar_t * scale,
                         const offset_t * size_out,
                         const offset_t * size_inp,
                         const offset_t * stride_out,
                         const offset_t * stride_inp)
{
    offset_t index = threadIdx.x + blockIdx.x * blockDim.x;
    offset_t nthreads = prod(size_out, ndim);

    for (offset_t i=index; index < nthreads;
         index += blockDim.x * gridDim.x, i=index)
    {
        offset_t x, y, z;
        offset_t batch_offset = index2offset_3d(i, ndim, size_out, stride_inp, x, y, z);
        offset_t out_offset = index2offset(i, ndim, size_out, stride_out);

        Multiscale<three, IX, BX, IY, BY, IZ, BZ>::resize(
            out + out_offset, inp + batch_offset,
            x, size_inp[ndim-3], stride_inp[ndim-3], scale[ndim-3],
            y, size_inp[ndim-2], stride_inp[ndim-2], scale[ndim-2],
            z, size_inp[ndim-1], stride_inp[ndim-1], scale[ndim-1],
            shift);
    }
}

template <int D, typename scalar_t, typename offset_t>
__global__ void kernelnd(scalar_t * out, scalar_t * inp, int ndim,
                         scalar_t shift, const scalar_t * scale,
                         const spline::type * order,
                         const bound::type * bnd,
                         const offset_t * size_out,
                         const offset_t * size_inp,
                         const offset_t * stride_out,
                         const offset_t * stride_inp)
{
    offset_t index = threadIdx.x + blockIdx.x * blockDim.x;
    offset_t nthreads = prod(size_out, ndim);

    for (offset_t i=index; index < nthreads;
         index += blockDim.x * gridDim.x, i=index)
    {
        offset_t x[D];
        offset_t batch_offset = index2offset_nd(i, ndim, size_out, stride_inp, x, D);
        offset_t out_offset = index2offset(i, ndim, size_out, stride_out);

        Multiscale<D>::resize(
            out + out_offset, inp + batch_offset,
            x, size_inp + ndim - D, stride_inp + ndim - D,
            order, bnd, scale, shift);
    }
}
