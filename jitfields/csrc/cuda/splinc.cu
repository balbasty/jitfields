#include "hip/hip_runtime.h"
#include "../lib/cuda_switch.h"
#include "../lib/splinc.h"
#include "../lib/bounds.h"
#include "../lib/batch.h"

using namespace jf;

template <bound::type B, typename scalar_t, typename offset_t>
__global__ void kernel(scalar_t * inp, int ndim,
                       const offset_t * size, const offset_t * stride,
                       const double * poles, int npoles)
{
    offset_t index = threadIdx.x + blockIdx.x * blockDim.x;
    offset_t nthreads = prod(size, ndim-1);

    for (offset_t i=index; index < nthreads;
         index += blockDim.x * gridDim.x, i=index)
    {
        offset_t offset = index2offset(i, ndim-1, size, stride);
        splinc::filter<B>(inp + offset, size[ndim-1], stride[ndim-1],
                          poles, npoles);
    }
}
